#include "hip/hip_runtime_api.h"

#include "gpuTools.hpp"
#include "Logger.hpp"
#include "Exceptions.hpp"



namespace microflow
{



void printDeviceProperties( hipDeviceProp_t devProp )
{
	double clockRateInGhz = devProp.clockRate / 1.0E6 ;
	double globalMemoryInMB = devProp.totalGlobalMem / (1024.0 * 1024.0) ; //FIXME: check for overflow

	logger << "Revision number           : " << devProp.major << "." ;
	logger << devProp.minor << "\n" ;
	logger << "Name                      : " << devProp.name  << "\n" ;
	logger << "Total global memory       : " << globalMemoryInMB << " MB\n" ;
	logger << "Clock rate                : " << clockRateInGhz << " GHz\n" ;
	logger << "Number of multiprocessors : " << devProp.multiProcessorCount << "\n" ;
	logger << "Kernel execution timeout  : " ;
	logger << (devProp.kernelExecTimeoutEnabled ? "Yes" : "No") << "\n" ;

	logger << "\n" ;
}



void initializeGPU( int gpuId )
{
	if (gpuId >= 0)
	{
		CUDA_CHECK( hipSetDevice(gpuId) ) ;
	}

	int i = -1 ;
	CUDA_CHECK( hipGetDevice(&i) ) ;

	logger << "\nUsing CUDA device #" << i << "\n" ;

	hipDeviceProp_t devProp;
	CUDA_CHECK(hipGetDeviceProperties(&devProp, i));
	printDeviceProperties(devProp);
}



void finalize()
{
	hipDeviceSynchronize() ;
	hipProfilerStop() ;
	hipDeviceReset() ;
}



void cudaCheck( hipError_t cudaCode, std::string file, size_t line )
{
  if ( hipSuccess != (cudaCode) )
  {
		std::stringstream sstr ;
		sstr << "Error at " << file << ":" << line 
    		 << " : " <<  hipGetErrorString(cudaCode) << "\n" ;
    THROW (sstr.str()) ;
  }
}



}
